#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <itpp/itbase.h>
#include <cpp/dev_random.cpp>
#include <cpp/itpp_ext_math.cpp>
#include <cpp/spinchain.cpp>
#include <math.h>
#include <tclap/CmdLine.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <tools.cpp>
#include "cuda_utils.cu"
#include "model.cu"
#include "ev_routines.cu"
#include "ev_math.cu"
#include "cuda_functions.cu"
#include "ex_model.cu"
#include <time.h>


TCLAP::CmdLine cmd("Command description message", ' ', "0.1");
// TCLAP::ValueArg<unsigned int> CseedArg("","Cseed", "Random seed [0 for urandom]",false, 0,"unsigned int",cmd);
TCLAP::ValueArg<unsigned int> EseedArg("","Eseed", "Random seed [0 for urandom]",false, 0,"unsigned int",cmd);
TCLAP::ValueArg<unsigned int> PARAMseedArg("","PARAMseed", "Random seed [0 for urandom]",false, 0,"unsigned int",cmd);
TCLAP::ValueArg<string> optionArg("o","option", "Option" ,false,"nichts", "string",cmd);
TCLAP::ValueArg<int> nqubitsArg("q","qubits", "Number of qubits",false, 3,"int",cmd);
TCLAP::ValueArg<int> numtArg("","t", "Number of time iterartions",false, 1,"int",cmd);
TCLAP::ValueArg<double> JArg("","Jc", "Ising interaction in the z-direction",false, 0.,"double",cmd);
TCLAP::ValueArg<double> JpArg("","Jp", "Ising interaction between A and B",false, 0.,"double",cmd);
TCLAP::ValueArg<double> DJs("","DJs", "Delta in the Js interacions on chain",false, 0.,"double",cmd);
TCLAP::ValueArg<double> Js("","Js", "Center of the Js interactions on chain",false, 0,"double",cmd);
TCLAP::ValueArg<double> bx("","bx", "Magnetic field in x direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> by("","by", "Magnetic field in y direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> bz("","bz", "Magnetic field in z direction",false, 0,"double",cmd);
TCLAP::ValueArg<int> kx("","kx", "Momentum field in x direction",false, 0,"int",cmd);
TCLAP::ValueArg<double> Dbs("","Dbs", "Delta in the magnetic field on spins",false, 0,"double",cmd);
TCLAP::ValueArg<int> one_state("","one_state", "State l",false, 0,"int",cmd);
TCLAP::ValueArg<int> ifrandom("","ifrandom", "0 if you dont want randstate",false,1,"int",cmd);
TCLAP::ValueArg<int> dev("","dev", "Gpu to be used, 0 for k20, 1 for c20",false, 0,"int",cmd);
TCLAP::SwitchArg no_general_report("","no_general_report","Print the general report", cmd);
TCLAP::ValueArg<string> modelArg("","model", "Option" ,false,"nichts", "string",cmd);
TCLAP::ValueArg<int> tAvg("","t_ave", "Number of time averaged over",false, 1,"int",cmd);
TCLAP::ValueArg<int> xlenArg("","x", "Some number x",false, 0,"int",cmd);
TCLAP::ValueArg<int> symr("","symR", "If symmetries sectors of reflections are used",false, 0,"int",cmd);

int main(int argc,char* argv[]) {
  // Set initial stuff
  cout.precision(17);
  hipSetDevice(dev.getValue());
  itpp::RNG_randomize();
  cmd.parse(argc,argv);
  string option=optionArg.getValue();
  string model=modelArg.getValue();
  double J=JArg.getValue();
  int nqubits = nqubitsArg.getValue();
  int numt=numtArg.getValue();
  int xlen=xlenArg.getValue();
  
  
  int l=pow(2,nqubits);    
  int nqubits_env,xl;
  
  itpp::ivec conxA,conxB;
  
  //Se elige el modelo a usar
  void (*evolution)(double *, double *, double, itpp::vec, int, int);
  evolution=model::lattice;
  nqubits_env=nqubits-1;

//   int Cseed=CseedArg.getValue();
  int PARAMseed=PARAMseedArg.getValue();int Eseed=EseedArg.getValue();
  
//   if (Cseed == 0 ){
//     Random seed_uran1; 
//     Cseed=seed_uran1.strong();
//   }
  itpp::RNG_reset(Cseed);
  //ESTADO INICIAL C
  itpp::cvec cstate = itppextmath::RandomState(2);
  
  if (Cseed == -1 ){
    cstate = itpp::ones_c(2);
    cstate=cstate*(1/sqrt(2));
  }
  
  if (Eseed == 0 ){
    Random seed_uran3; 
    Eseed=seed_uran3.strong();
  }
  itpp::RNG_reset(Eseed);
  
  itpp::cvec state;
  if(xlen==0) {
    itpp::cvec estate = itppextmath::RandomState(l/2);
  
    //Preparacion estado inicial
    state=tensor_prod(cstate,estate);
  }
  else {
    xl=pow(2,xlen);
    itpp::cvec estateA = itppextmath::RandomState(xl);
    itpp::cvec estateB = itppextmath::RandomState(l/(xl*2));
  
    //Preparacion estado inicial
    state=tensor_prod(cstate,tensor_prod(estateB,estateA)); 
  }
  
  //PARAMETROS SEED
  if (PARAMseed == 0 ){
    Random seed_uran2; 
    PARAMseed=seed_uran2.strong();
  }
  itpp::RNG_reset(PARAMseed);
  itpp::vec js = itpp::ones(nqubits_env)*(Js.getValue()-DJs.getValue()) + itpp::randu(nqubits_env)*(2*DJs.getValue());

  // Campo magnético
  itpp::vec b_one(3); b_one(0)=bx.getValue(); b_one(1)=by.getValue(); b_one(2)=bz.getValue();
  itpp::vec b(3);
  b(0)=b_one(0);
  b(1)=0;
  b(2)=b_one(2);
  
  //Se sube el estado al dev
  double *dev_R,*dev_I;
  evcuda::itpp2cuda_malloc(state,&dev_R,&dev_I);

  // Se calculan las trazas
  itpp::cvec stateBra=state;
  for (int t = 0; t < numt; t++) {
    evolution(dev_R,dev_I,J,b,nqubits,xlen);
    evcuda::cuda2itpp(state,dev_R,dev_I);
    cout<<norm(itpp::dot(itpp::conj(stateBra),state))<<endl;
  }
}
