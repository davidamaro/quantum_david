#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <itpp/itbase.h>
#include <cpp/dev_random.cpp>
#include <cpp/itpp_ext_math.cpp>
#include <cpp/spinchain.cpp>
#include <math.h>
#include <tclap/CmdLine.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include "tools.cpp"
#include "cuda_utils.cu"
#include "model.cu"
#include "ev_routines.cu"
#include "ev_math.cu"
#include "cuda_functions.cu"
#include "ex_model.cu"
#include <time.h>


TCLAP::CmdLine cmd("Command description message", ' ', "0.1");
TCLAP::ValueArg<unsigned int> CseedArg("","Cseed", "Random seed [0 for urandom]",false, 0,"unsigned int",cmd);
TCLAP::ValueArg<unsigned int> EseedArg("","Eseed", "Random seed [0 for urandom]",false, 0,"unsigned int",cmd);
TCLAP::ValueArg<unsigned int> PARAMseedArg("","PARAMseed", "Random seed [0 for urandom]",false, 0,"unsigned int",cmd);
TCLAP::ValueArg<int> nqubitsArg("q","qubits", "Number of qubits",false, 3,"int",cmd);
TCLAP::ValueArg<int> numtArg("","t", "Number of time iterartions",false, 1,"int",cmd);
TCLAP::ValueArg<double> JArg("","Jc", "Ising interaction in the z-direction",false, 0.,"double",cmd);
TCLAP::ValueArg<double> bx("","bx", "Magnetic field in x direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> by("","by", "Magnetic field in y direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> bz("","bz", "Magnetic field in z direction",false, 0,"double",cmd);
TCLAP::ValueArg<int> kx("","kx", "Momentum field in x direction",false, 0,"int",cmd);
TCLAP::ValueArg<int> dev("","dev", "Gpu to be used, 0 for k20, 1 for c20",false, 0,"int",cmd);
TCLAP::ValueArg<string> modelArg("","model", "Option" ,false,"nichts", "string",cmd);
TCLAP::ValueArg<int> xlenArg("","x", "Some number x",false, 0,"int",cmd);

int main(int argc,char* argv[]) {
  // Set initial stuff
  cout.precision(17);
  hipSetDevice(dev.getValue());
  cmd.parse(argc,argv);
  string model=modelArg.getValue();
  double J=JArg.getValue();
  int nqubits = nqubitsArg.getValue();
  int numt=numtArg.getValue();
  int xlen=xlenArg.getValue();
  int l=pow(2,nqubits);    
  int xl;

  //Se elige el modelo a usar
  void (*evolution)(double *, double *, double, itpp::vec, int, int);
  evolution=model::lattice;

  int Cseed=CseedArg.getValue();int PARAMseed=PARAMseedArg.getValue();int Eseed=EseedArg.getValue();
  
  if (Cseed == 0 ){
    Random seed_uran1; 
    Cseed=seed_uran1.strong();
  }
  itpp::RNG_reset(Cseed);
  //ESTADO INICIAL C
  itpp::cvec cstate = itppextmath::RandomState(2);
  
  if (Cseed == -1 ){
    cstate = itpp::ones_c(2);
    cstate=cstate*(1/sqrt(2));
  }
  
  if (Eseed == 0 ){
    Random seed_uran3; 
    Eseed=seed_uran3.strong();
  }
  itpp::RNG_reset(Eseed);
  
  itpp::cvec state;
  if(xlen==0) {
    itpp::cvec estate = itppextmath::RandomState(l/2);
  
    //Preparacion estado inicial
    state=tensor_prod(cstate,estate);
  }
  else {
    xl=pow(2,xlen);
    itpp::cvec estateA = itppextmath::RandomState(xl);
    itpp::cvec estateB = itppextmath::RandomState(l/(xl*2));
  
    //Preparacion estado inicial
    state=tensor_prod(cstate,tensor_prod(estateB,estateA)); 
  }

  // Campo magnético
  itpp::vec b_one(3); b_one(0)=bx.getValue(); b_one(1)=by.getValue(); b_one(2)=bz.getValue();
  itpp::vec b(3);
  b(0)=b_one(0);
  b(1)=0;
  b(2)=b_one(2);
  
  //Se sube el estado al dev
  double *dev_R,*dev_I;
  evcuda::itpp2cuda_malloc(state,&dev_R,&dev_I);

  // Se calculan las trazas
  itpp::cvec stateBra=state;
  for (int t = 0; t < numt; t++) {
    evolution(dev_R,dev_I,J,b,nqubits,xlen);
    evcuda::cuda2itpp(state,dev_R,dev_I);
    cout<<norm(itpp::dot(itpp::conj(stateBra),state))<<endl;
  }
}
